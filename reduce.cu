#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "utils.cpp"
#include "hip/hip_runtime.h"
#include "nccl.h"

using namespace std;

Clock tim;

typedef double db;

const int threadsPerBlock=256;

const int device_cnt=8;
int data_len=1<<20;
int virt_dev[device_cnt];

inline bool db_same(db a,db b){
	return fabs(a-b)<1e-9;
}

inline void get_devices(){
	int cnt=0;
	hipGetDeviceCount(&cnt);
	assert(cnt>0);
	cout<<"Device list:{";
	for(int i=0;i<device_cnt;i++){
		virt_dev[i]=i%cnt;
		cout<<virt_dev[i]<<(i<device_cnt-1?",":"}");
	}
	cout<<endl;
}

double *inputs[device_cnt];

double *cpu_output;
double *gpu_output;
double *nccl_output;

inline void get_data(){
	for(int i=0;i<device_cnt;i++){
		inputs[i]=new db[data_len];
		for(int j=0;j<data_len;j++){
			static uniform_real_distribution<db>rnd(0,1);
			static default_random_engine eng;
			inputs[i][j]=rnd(eng);
		}
	}
	cout<<"get_data complete"<<endl;
}

inline void run_cpu(){
	cpu_output=new db[data_len];
	fill_n(cpu_output,data_len,0);
	tim.tic();
	for(int i=0;i<device_cnt;i++){
		for(int j=0;j<data_len;j++){
			cpu_output[j]+=inputs[i][j];
		}
	}
	cout<<"run_cpu:"<<tim.toc()<<endl;
}

__global__ void doAdd(db* inputs,db* out,int data_len){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	db sum=0;
	for(int j=0;j<device_cnt;j++){
		sum+=inputs[j*data_len+i];
	}
	out[i]=sum;
}

inline void run_gpu(){
	db *d_input[device_cnt];
	hipStream_t stream[device_cnt];
	const int root=0;
	db *gather,*dest;
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamCreate(&stream[i]);
		hipMalloc(&d_input[i],data_len*sizeof(db));
		hipMemcpyAsync(d_input[i],inputs[i],data_len*sizeof(db),hipMemcpyHostToDevice,stream[i]);
	}
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamSynchronize(stream[i]);
	}
	hipSetDevice(virt_dev[root]);
	hipMalloc(&gather,data_len*device_cnt*sizeof(db));
	hipMalloc(&dest,data_len*sizeof(db));
	tim.tic();
	for(int i=0;i<device_cnt;i++){
		hipMemcpyAsync(gather+i*data_len,d_input[i],data_len*sizeof(db),hipMemcpyDeviceToDevice,stream[i]);
	}
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamSynchronize(stream[i]);
	}
	hipSetDevice(virt_dev[root]);
	doAdd<<<data_len/threadsPerBlock,threadsPerBlock>>>(gather,dest,data_len);
	cout<<"run_gpu:"<<tim.toc()<<endl;
	gpu_output=new db[data_len];
	hipMemcpy(gpu_output,dest,data_len*sizeof(db),hipMemcpyDeviceToHost);
	hipFree(dest);
	hipFree(gather);
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipFree(d_input[i]);
		hipStreamDestroy(stream[i]);
	}
	cout<<"\tchecking gpu.."<<endl;
	for(int i=0;i<data_len;i++){
		if(!db_same(gpu_output[i],cpu_output[i])){
			cout<<"i="<<i<<"\tgpu:"<<gpu_output[i]<<"\tcpu:"<<cpu_output[i]<<endl;
		}
		assert(db_same(gpu_output[i],cpu_output[i]));
	}
	cout<<"\tpassed"<<endl;
}

inline void run_nccl(){
	db *d_input[device_cnt];
	hipStream_t stream[device_cnt];
	ncclComm_t comm[device_cnt];
	ncclUniqueId id;
	ncclGetUniqueId(&id);
	ncclGroupStart();
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamCreate(&stream[i]);
		hipMalloc(&d_input[i],data_len*sizeof(db));
		hipMemcpyAsync(d_input[i],inputs[i],data_len*sizeof(db),hipMemcpyHostToDevice,stream[i]);
		ncclCommInitRank(&comm[i],device_cnt,id,i);
	}
	ncclGroupEnd();
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamSynchronize(stream[i]);
	}
	//data is ready on gpu
	const int root=0;
	hipSetDevice(virt_dev[root]);
	db *dest;
	hipMalloc(&dest,data_len*sizeof(db));
	tim.tic();
	ncclGroupStart();
	for(int i=0;i<device_cnt;i++){
		ncclReduce(d_input[i],dest,data_len,ncclDouble,ncclSum,root,comm[i],stream[i]);
	}
	ncclGroupEnd();
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamSynchronize(stream[i]);
	}
	cout<<"run_nccl:"<<tim.toc()<<endl;
	hipSetDevice(virt_dev[root]);
	nccl_output=new db[data_len];
	hipMemcpy(nccl_output,dest,data_len*sizeof(db),hipMemcpyDeviceToHost);
	hipFree(dest);
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipFree(d_input[i]);
		hipStreamDestroy(stream[i]);
		ncclCommDestroy(comm[i]);
	}
	cout<<"\tchecking nccl..."<<endl;
	for(int i=0;i<data_len;i++){
		assert(db_same(nccl_output[i],cpu_output[i]));
	}
	cout<<"\tpassed"<<endl;
}

int main(){
	cout<<"data_len exp:";
	cin>>data_len;
	data_len=1<<data_len;
	get_devices();
	get_data();
	run_cpu();
	run_gpu();
	run_nccl();
}
