#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "utils.cpp"
#include "hip/hip_runtime.h"
#include "nccl.h"

Clock tim;

typedef double db;

const int device_cnt=8;
const int data_len=1e7;
int virt_dev[device_cnt];

inline void get_devices(){
	int cnt=0;
	hipGetDeviceCount(&cnt);
	assert(cnt>0);
	cout<<"Device list:{";
	for(int i=0;i<device_cnt;i++){
		virt_dev[i]=i%cnt;
		cout<<virt_dev[i]<<i<device_cnt-1?",":"}";
	}
	cout<<endl;
}

double *inputs[device_cnt];

double *cpu_output;
double *nccl_output;

inline void get_data(){
	for(int i=0;i<device_cnt;i++){
		inputs[i]=new db[data_len];
		for(int j=0;j<data_len;j++){
			static uniform_real_distribution<db>rnd(0,1);
			static default_random_engine eng;
			inputs[i][j]=rnd(eng);
		}
	}
	cout<<"get_data complete"<<endl;
}

inline void run_cpu(){
	cpu_output=new db[data_len];
	fill_n(cpu_output,n,0);
	tim.tic();
	for(int i=0;i<device_cnt;i++){
		for(int j=0;j<data_len;j++){
			cpu_output[j]+=inputs[i][j];
		}
	}
	cout<<"run_cpu:"<<tim.tok()<<endl;
}

__global__ doAdd(db* inputs,db* out){
	int i=blockDim.x*blockIdx.x;
	db sum=0;
	for(int j=0;j<device_cnt;j++){
		sum+=inputs[j*device_cnt+i];
	}
	out[i]=sum;
}

inline void run_gpu(){
	db *d_input[device_cnt];
	hipStream_t stream[device_cnt];
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamCreate(&stream[i]);
		hipMalloc(&d_input[i],data_len*sizeof(db));
		hipMemcpy(d_input[i],input[i],data_len*sizeof(db),hipMemcpyHostToDevice,stream[i]);
	}
	tim.tic();
	int root=0;
	hipSetDevice(virt_dev[0]);
	db *gather,*dest;
	hipMalloc(&gather,data_len*device_cnt*sizeof(db));
	for(int i=0;i<device_cnt;i++){
		hipMemcpy(gather+i*data_len,d_input[i],data_len*sizeof(db),hipMemcpyDeviceToDevice,stream[i]);
	}
	for(int i=0;i<device_cnt;i++){
		hipStreamSynchronize(stream[i]);
	}
	hipMalloc(&dest,data_len*sizeof(db));
	const int threadsPerBlock=256;
	data_len<<<data_len/threadsPerBlock,threadsPerBlock>>>(gather,dest);
	cout<<"run_gpu:"<<tim.tok()<<endl;
	gpu_output=new db[data_len];
	hipMemcpy(gpu_output,dest,data_len*sizeof(db),hipMemcpyDeviceToHost);
	hipFree(dest);
	hipFree(gather);
	for(int i=0;i<device_cnt;i++){
		hipFree(d_input[i]);
		hipStreamDestroy(stream[i]);
	}
}

inline void run_nccl(){
	db *d_input[device_cnt];
	hipStream_t stream[device_cnt];
	ncclComm_t comm[device_cnt];
	ncclUniqueId id;
	ncclGetUniqueId(&id);
	ncclGroupStart();
	for(int i=0;i<device_cnt;i++){
		hipSetDevice(virt_dev[i]);
		hipStreamCreate(&stream[i]);
		hipMalloc(&d_input[i],data_len*sizeof(db));
		hipMemcpy(d_input[i],input,data_len*sizeof(db),hipMemcpyHostToDevice,stream[i]);
		ncclCommInitRank(comm+i,device_cnt,id,i);
	}
	ncclGroupEnd();
	for(int i=0;i<device_cnt;i++){
		hipStreamSynchronize(stream[i]);
	}
	//data is ready on gpu
	const int root=0;
	hipSetDevice(virt_dev[root]);
	db *dest;
	hipMalloc(&dest,data_len*sizeof(db));
	tim.tic();
	ncclGroupStart();
	for(int i=0;i<device_cnt;i++){
		ncclReduce(d_input[i],dest,data_len,ncclDouble,ncclSum,root,comm[i],stream[i]);
	}
	ncclGroupEnd();
	for(int i=0;i<device_cnt;i++){
		hipStreamSynchronize(stream[i]);
	}
	cout<<"run_nccl:"<<tim.tok()<<endl;
	gpu_output=new db[data_len];
	hipMemcpy(gpu_output,dest,data_len*sizeof(db));
	for(int i=0;i<device_cnt;i++){
		hipFree(d_input[i]);
		hipStreamDestroy(stream[i]);
	}
	hipFree(dest);
}

inline bool same(db a,db b){
	return fabs(a-b)<1e-9;
}

int main(){
	get_devices();
	get_data();
	run_cpu();
	run_gpu();
	run_nccl();
	cout<<"starting check..."<<endl;
	for(int i=0;i<data_len;i++){
		assert(db_same(gpu_output[i],cpu_output[i]));
		assert(db_same(nccl_output[i],cpu_output[i]));
	}
	cout<<"check complete."<<endl;
}
